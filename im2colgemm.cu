#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <sstream>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
// Feature maps dimensionality descriptions and assumptions:
//             : Height          : Width           : Channels  : Number                    :
// INPUT   / A | H               | W               | C         | ------------------------- |  
// KERNELS / F | P = K           | Q = K           | R = C     | D = number of kernels = 1 |
// OUTPUT  / B | L = H * (K - 1) | M = W * (K - 1) | N = D = 1 | ------------------------- |
// [!] K must be odd number.
// [!] Data layout for INPUT/OUTPUT: C x H x W.
// [!] Data layout for KERNELS: D x R(=C) x P(=K) x Q(=K)

// Turn on/off debug mode
 #define DEBUG
 #define FUNCTEST
#define PERFTEST

#ifdef DEBUG
    #define LOG(...) printf(__VA_ARGS__); fflush(stdout);
#else
    #define LOG(...) ;
#endif

const unsigned int H = 100, W = 100, C = 100, K = 3, maxDilation=4, C_out=80; 

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
    
    //Namespace for std
    using namespace std;

    //structure declaration for storing rows and columns for a matrix
    struct matrix{
        unsigned int rows;  //storing rows of a matrix
        unsigned int cols;  //storing columns of a matrix
    };

    //handlerror declaration : to display file and line numbers of erroneous lines
    static void HandleError( hipError_t err, const char *file, int line ) {
        if (err != hipSuccess) {
            cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
        }
    }

    //handle error alias name declaration
    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


    //global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
    //matrix multiplication is being done tile by tile
    __global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
    {   
        //shared memory takes one tile at a time
        __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];   //to store tiles for array 1
        __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];   //to store tiles for array 2

        //threads x and y index for the current block
        unsigned int tx=threadIdx.x;    
        unsigned int ty=threadIdx.y;

        unsigned int c=blockIdx.x*blockDim.x + threadIdx.x; //row value using x-index of current thread
        unsigned int r=blockIdx.y*blockDim.y + threadIdx.y; //column value using y-index of current thread

        unsigned int idx=c*rows1+r;             //column major index, using row and column value
        
        float val=0;        //register to store multiplication result initialized to zero

        for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)    //going over all tiles one by one, with each m
        {

            int var1=m*TILE_WIDTH+tx ;      //x thread value for current tile
            int var2=m*TILE_WIDTH+ty ;      //y thread value for current tile
            
            //copying a tile from array1
            if (r < rows1 && var1 < rows2)      //if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
                S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
            else
                    S1[ty][tx]=0;                   //storing zero, since there is no valid value
                __syncthreads();                        //syncing all threads once shared memory S1 is stored
            
            //copying a tile from array2
                if(c < cols2 && var2 < rows2)   //if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
                    S2[ty][tx]=array2[var2+rows2*c];    //storing the valid value
                else 
                    S2[ty][tx]=0;       //storing zero, since no valid value
            __syncthreads();        //synchronizing threads
            

            for(int i=0; i<TILE_WIDTH;i++)  //going over entire tile, ty row in S1 and tx column in S2
                val+=S1[ty][i]*S2[i][tx];   //and multiplying elements
            __syncthreads();        //synchronizing threads

        }
        
        if(r < rows1 && c< cols2)   //removing degenerate cases
            array3[idx]=val;    //saving multiplication result to global memory
            
    }

    float* gemm(float *array_A, float *array_B, int M_Arows, int M_Acols, int M_Brows, int M_Bcols) {

        float* array_C=(float*)malloc(M_Arows*M_Bcols*sizeof(float));//array to store gpu result in column major format
        
        float* array_D=(float*)malloc(M_Arows*M_Bcols*sizeof(float));//arary to store cublas result in column major format

        //GPU DEVICE PROPERTIES and selecting a GPU for calculation
        int nDevices;
        HANDLE_ERROR(hipGetDeviceCount(&nDevices));

        hipDeviceProp_t prop;
        HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));    //using GPU0

        //BLOCK AND GRID SIZE DECLARATION
        float thread_block=sqrt(prop.maxThreadsPerBlock);   //2D blocks used
        dim3 DimGrid(ceil(M_Bcols/thread_block),ceil(M_Arows/thread_block),1); //image saved as a 2D grid
        dim3 DimBlock(thread_block,thread_block,1);

        size_t Sbytes = 2* DimBlock.x * DimBlock.y ;    //2 arrays used in the calculation, hence 2 * DimBlock.x * DimBlock.y
        
        //Checking if sufficient shared memory available or not

        if(prop.sharedMemPerBlock < Sbytes){
            std::cout<<"ERROR: insufficient shared memory"<<std::endl;
            exit(1);
        }

        //GPU MEMORY ALLOCATION
        float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;   //gpu arrays declared
       
        HANDLE_ERROR(hipMalloc(&array_A_gpu,M_Arows*M_Acols*sizeof(float))); //allocate space to store arrayA

        HANDLE_ERROR(hipMalloc(&array_B_gpu,M_Brows*M_Bcols*sizeof(float))); //allocate space to store arrayB

        HANDLE_ERROR(hipMalloc(&array_C_gpu,M_Arows*M_Bcols*sizeof(float))); //allocate space to store gpu result

        HANDLE_ERROR(hipMalloc(&array_D_gpu,M_Arows*M_Bcols*sizeof(float))); //allocate space to store cublas result


        //COPY TO GPU MEMORY
        HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_Arows*M_Acols*sizeof(float), hipMemcpyHostToDevice));//copy arrayA to gpu

        HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_Brows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayB to gpu

        HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_Arows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayC to gpu

        HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_Arows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayD to gpu


        //time measurement for matrix multiplication
        hipEvent_t start1, stop1;
        
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
        
        //MATRIX MULTIPLICATION USING KERNEL
        hipEventRecord(start1);
        matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_Arows,M_Acols,array_B_gpu,M_Brows,M_Bcols,array_C_gpu);//calling the kernel
        hipEventRecord(stop1);

        hipEventSynchronize(stop1);

        float milliseconds1 = 0, milliseconds2 = 0;//storing the execution time in milliseconds
        
        hipEventElapsedTime(&milliseconds1, start1, stop1);//get the time in milliseconds
        cout<<"Time taken by GPU GEMM = "<<milliseconds1<<" ms"<<endl;//printing time taken by GPU

        //copy to CPU MEMORY
        HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_Arows*M_Bcols*sizeof(float), hipMemcpyDeviceToHost));//copying result of multiplication from gpu to cpu

        //Creating handle for CUBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);  

        //parameter declaration for cublas implementation
        float alpha = 1.0;
        float beta = 0.0;
        
        //cublas time measurement
        hipEvent_t start2, stop2;
        
        hipEventCreate(&start2);
        hipEventCreate(&stop2);

        //MATRIX MULTIPLICATION USING CUBLAS 
        hipEventRecord(start2);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_Arows, M_Bcols, M_Acols, &alpha, array_A_gpu, M_Arows, array_B_gpu, M_Brows, &beta, array_D_gpu, M_Arows);
        hipEventRecord(stop2);

        hipEventSynchronize(stop2);

        hipEventElapsedTime(&milliseconds2, start2, stop2);//get the time in milliseconds
        cout<<"Time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;//printing time taken by CUBLAS
        
        //copy to CPU MEMORY
        
            HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_Arows*M_Bcols*sizeof(float), hipMemcpyDeviceToHost));//copy result of multiplication using CUBLAS from gpu to cpu

        //CALCULATING MEAN SQUARED ERROR IN BOTH METHODS OF MATRIX MULTIPLICATION
        float mse=0; //mean squared error;

     /*   for(int i=0; i<M_Arows*M_Bcols;i++) {
            mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);//calculating element by element
           // printf("%.3f ", array_C[i]);
        }
        mse=mse/(M_Arows*M_Bcols);  //taking the mean of squared error
            
        cout<<endl<<"Mean square error = "<<mse<<endl;//printing out the mean squared error*/
	return array_C;

    }


float* flatten_kernel(float * weights, int k, int d, int c_rows){
    int c_cols = (k + (k-1)*(d-1))*(k + (k-1)*(d-1));
    float *canvas=(float*)calloc(c_rows*c_cols,sizeof(float));
    int itr = 0;
    int k_id = 0;
    for(int dilation = 1; dilation<=d; dilation++){
    int cur_kernel_size = k + (k-1)*(dilation-1);
    for(int kernel_id = 0; kernel_id <1; kernel_id++){
    itr = k_id*c_cols + (d-dilation)* pow(c_cols,0.5) + (d - dilation);
    for(int weight_id = 0; weight_id < k*k*C; weight_id++){
    canvas[itr] = weights[k_id*k*k + weight_id];
   /* cout<< weights[k_id*k*k+weight_id]<<" WEIGHTS"<<endl;
    cout<<"  k_id  "<<k_id<<endl;
    cout<<k_id*k*k+weight_id<<"    index    "<<endl;
    cout<<canvas[itr]<<" CANVAS "<<itr <<endl;*/
    itr++;	
    if(((k_id*k*k + weight_id)+1)%(k)==0){
    	for(int last_col_pads = 0; last_col_pads<(dilation-1)*(pow(c_cols,0.5)) + (pow(c_cols,0.5)-(cur_kernel_size  ));last_col_pads++ ){
    		//canvas[itr] = 0;
    		//cout<<itr<< "TEST" <<endl;
    		itr++;
    	
    	}
    }
    else{
    	
    	for(int inner_cols = 0; inner_cols<(dilation-1);inner_cols++ ){
    		//canvas[itr] = 0;
    		itr++;
    	}

    }

    }
    k_id++;
    }
    }
    float *canvas_col=(float*)calloc(c_rows*c_cols,sizeof(float));
    itr=0;
    for(int i=0; i<c_cols;i++){
    for(int j=0; j < c_rows; j++) {
    canvas_col[itr]=canvas[(j*c_cols)+i];
    itr++;
    }
    }

return canvas_col;
	
}

 
// DEVICE KERNEL
// Takes matrix A [float *matA] and transforms it
// into column representation [float *matAc] on GPU
__global__ 
void im2colOnDevice(unsigned int n, float *matAc, float *matA, int H_, int W_, int L, int M, int K, int C)
{
   // Using grid-stride loop if too big problem size.
    // https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < n; 
         idx += blockDim.x * gridDim.x) 
    {
        int m = (idx/C ) / L;
        int l = (idx/C ) % L;
        int r = idx % C;
        
        // For each spatial position in output...
        if (m < M) {
            int w = m ;
            if (l < L) {
                int h = l;
                // For each kernel weight...
                for (int x = 0; x < K; x++) {
        
            h = l;
                    for (int y = 0; y < K; y++) {
                        if (r < C) {
                            matAc[(idx*K*K) + (x*K + y)] = matA[(r*(H_*W_)+(w*W_ + h ))]; 
            h++;                        
}
                    }w++;
                }
            }
        }
    }
}
 
float* padding(unsigned int maxDilation, float* matInput) {
    int newH=H+(2*maxDilation);
    int newW=W+(2*maxDilation);
    //const size_t size\ = newH*newW*sizeof(float);
    float *paddedInput=(float *)calloc(newH*newW, sizeof(float));
    for(int x=0; x<H;x++) {
        for(int y=0; y<W;y++) {
            paddedInput[((x+maxDilation)*newW)+y+maxDilation]=matInput[(x*W) + y];
        }

    }
    return paddedInput;

}


void program(unsigned int blockSize, unsigned int gridSize = 0)
{
    // CONSTS AND VARIABLES
    // Input/kernel/output counts and sizes
    const unsigned int countA = H*W*C;
    const size_t sizeA = countA*sizeof(float);
//    LOG("[i] INPUT PARAMS: %u height, %u width, %u channels, %u elems, %u bytes\n", H, W, C, countA, sizeA);

    const unsigned int countF = K*K*C;
    const size_t sizeF = countF*sizeof(float);
  //  LOG("[i] FILTER PARAMS: %u elems, %u bytes\n", countF, countF*sizeof(float));
 int paddedH=H+(2*maxDilation);
    int paddedW=W+(2*maxDilation);    
    const unsigned int L = H;
    const unsigned int M = W;
    const unsigned int KERNELS=L*M*C;	

    //LOG("[i] OUTPUT PARAMS: %u height, %u width, %u channels\n", L, M, 1);
    
    //dilated kernel size
    int K_= K + (K-1)*(maxDilation-1);
    const unsigned int countF_ = K_*K_*C;
    const unsigned int countLR = L * M;
    const unsigned int countAc = countF_ * countLR;
    const size_t sizeAc = countAc*sizeof(float);
    //LOG("[i] INPUT IN COL PARAMS: %u elems, %u bytes\n", countAc, sizeAc);
	
    const unsigned int countKc = K_*K_*C_out;
    
    // PREPARE DATA

    // Generate input data
    float *matA = (float *)malloc(sizeA);
    //float matA[36] = {0.0,0.0,0.0,0.0,0.0,0.0,0.0,1.0,2.0,3.0,4.0,0.0,0.0,5.0,6.0,7.0,8.0,0.0,0.0,9.0,10.0,11.0,12.0,0.0,0.0,13.0,14.0,15.0,16.0,0.0};
    for (int i = 0; i < countA ; i++) {
        matA[i] =(float)(i+1);
    //printf("%.1f ",matA[i]);
   // if((i+1)%W==0){
    //printf("\n");}
    }
    //printf("\n");	
    //LOG("[i] PADDED INPUT PARAMS: %u height, %u width \n", paddedH, paddedW);
    float *matInput=padding(maxDilation, matA);
   // for(int i=0; i<paddedH; i++) {
     //   for(int j=0; j < paddedW; j++) {
       //     int index=j+(i*paddedW);
         //   printf("%.1f ",matInput[index]);
        //if((index+1)%(W+(2*maxDilation))==0){
        //printf("\n");}
        //}
    //}
   // LOG("  [!] FINISHED GENERATING INPUT\n");*/
    // Alloc memory and copy data to device
    float *devA, *devAc, *retAc;
    const size_t sizeI = paddedW*paddedH*sizeof(float);
    hipMalloc((void**)&devA, sizeI); 
    hipMalloc((void**)&devAc, sizeAc); 
    retAc = (float *)malloc(sizeAc);

    hipMemcpy(devA, matInput, sizeI, hipMemcpyHostToDevice); 

    // Compute default grid size if it wasn't passed
    const unsigned int KERNELS_NUM = L * M * C;
    if (gridSize == 0)
        gridSize = (KERNELS_NUM + blockSize - 1) / blockSize;
    
    // Run im2col computation on device and copy results
    struct timeval t3, t4;
    gettimeofday(&t3, NULL);
    im2colOnDevice<<<100, 1000>>>(KERNELS, devAc, devA, paddedH, paddedW, L, M, K_ , C);
	    gettimeofday(&t4, NULL);
    LOG("  [!] FINISHED CALCULATING im2col ON DEVICE %.16fms\n",(t4.tv_usec-t3.tv_usec)/1000.0+(t4.tv_sec-t3.tv_sec)*1000.0);    
    hipMemcpy(retAc, devAc, sizeAc, hipMemcpyDeviceToHost);
//    for (int i = 0; i < countAc; i++) {
  //      printf("%.1f ",retAc[i]);
   //     if((i+1)%K_ == 0)
    //    printf("\n");
     //       if((i+1)%(K_*K_)==0)
      //  {printf("\n\n\n");}
       //     }
   // printf("\n");

//GEMM
float *matFlatten = (float *)malloc(sizeF*C_out);
for (int i = 0; i < countF*C_out; i++) {
        matFlatten[i] =(float)(1);
    }
//printf("KERNEL MATRIIX \n");
struct timeval flattens, flattene;
    gettimeofday(&flattens, NULL);
float* kernelMatrix=flatten_kernel(matFlatten,K, maxDilation, C_out);
gettimeofday(&flattene, NULL);
 LOG("  [!] FINISHED CALCULATING Flatten ON DEVICE %.16fms\n",(flattene.tv_usec-flattens.tv_usec)/1000.0+(flattene.tv_sec-flattens.tv_sec)*1000.0);
//printf("\n\n");
//for(int i=0; i<countKc;i++) {
//	printf("%f  ",kernelMatrix[i]);
//	if((i+1)%(K_*K_)==0)
//		printf("\n\n");
//
//}
//printf("\n\n");

//TODO: CHECK COUNTLR
float *res_gemm = gemm(kernelMatrix, retAc, C_out, countF_, K_ *K_ *C, countLR);
    gettimeofday(&t4, NULL);
    LOG("  [!] FINISHED CALCULATING CoConv ON DEVICE %.16fms\n",(t4.tv_usec-t3.tv_usec)/1000.0+(t4.tv_sec-t3.tv_sec)*1000.0);
//for(int i=0; i<C_out*countLR;i++) {
  //  printf("%.3f ", res_gemm[i]);
//}
//printf("\n\n");
/*for(int c=0; c<C_out; c++){
int spaces=0;
for (int i = 0; i < countLR; i++) {
spaces++;
int idx=i*C_out  + (c);
        printf("%.1f ",res_gemm[idx]);
        if((spaces)%L == 0)
         	printf("\n");
    }
printf("\n\n");
}
printf("\n");*/
    // CLEAN UP
    hipFree(devA);
    hipFree(devAc);
    
    //free(matA);
    //free(matInput);
    //free(matFlatten);
    //free(retAc);
}

int main()
{
    // Enforce default block and grid sizes
    unsigned int blockSize = 256;
    unsigned int gridSize = 0;

    // Calculate max needed kernels/threads number
    const unsigned int L = H - (K - 1);
    const unsigned int M = W - (K - 1);
    const unsigned int KERNELS_NUM = L * M * C;

    // Prepare variables for time measurement
    struct timeval t1, t2;
    double elapsedTime, totalTime = 0;
    int totalRuns = 1;
    
    // First warm-up run
   // LOG("--------- WARM-UP ---------\n");
    //program(256);
    //LOG("--------- WARM-UP ---------\n\n");

#ifdef PERFTEST
    // Average over 10 runs
    totalRuns = 1;
    
    // Open file for perf logs
    std::fstream fperflog("perflog.csv", std::ios::out);
    if (fperflog.good())
    {
        // Measure effect of different block sizes
        const unsigned int MAX_BLOCK_SIZE = 2048;
        for (blockSize = 32; blockSize <= 34; blockSize *= 2) {
            const unsigned int MAX_GRID_SIZE = (KERNELS_NUM + blockSize - 1) / blockSize;
            LOG("  [!] For %d blocks, max grid size is %d\n", blockSize, MAX_GRID_SIZE);
            for (gridSize = 1; gridSize <= 1; gridSize *= 2) {
                if (gridSize <= MAX_GRID_SIZE) {
                    totalTime = 0;
                    for (int i = 0; i < 1; i++)
#endif
                    {
                        // Start timer
                        gettimeofday(&t1, NULL);
                    
                        // WORK HARD!
                        program(blockSize, gridSize);
                    
                        // Stop timer
                        gettimeofday(&t2, NULL);
                    
                        // Compute the elapsed time in millisec
                        elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
                        elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
                    
                        totalTime += elapsedTime;
                    }
                    LOG("  [!] Whole program took %.3fms averaged over %d runs\n", totalTime / totalRuns, totalRuns);
#ifdef PERFTEST
                    fperflog << blockSize << "," << gridSize << "," << elapsedTime << std::endl;
                } else {
                    // Meaningless data, there is more grids ten data cat utilize 
                    fperflog << blockSize << "," << gridSize << "," << -1 << std::endl;
                }
            }
        }
        
        // Close file
        fperflog.close();
    }
#endif

    return EXIT_SUCCESS;
}


