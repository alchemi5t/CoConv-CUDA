#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <sstream>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <unistd.h>
#include <string.h>
// Feature maps dimensionality descriptions and assumptions:
//             : Height          : Width           : Channels  : Number                    :
// INPUT   / A | H               | W               | C         | ------------------------- |  
// KERNELS / F | P = K           | Q = K           | R = C     | D = number of kernels = 1 |
// OUTPUT  / B | L = H * (K - 1) | M = W * (K - 1) | N = D = 1 | ------------------------- |
// [!] K must be odd number.
// [!] Data layout for INPUT/OUTPUT: C x H x W.
// [!] Data layout for KERNELS: D x R(=C) x P(=K) x Q(=K)

// Turn on/off debug mode
 #define DEBUG
 #define FUNCTEST
#define PERFTEST

#ifdef DEBUG
    #define LOG(...) printf(__VA_ARGS__); fflush(stdout);
#else
    #define LOG(...) ;
#endif

unsigned int H, W, C = 100, K = 3, maxDilation=4, C_out=80; 

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
    
    //Namespace for std
    using namespace std;

    //structure declaration for storing rows and columns for a matrix
    struct matrix{
        unsigned int rows;  //storing rows of a matrix
        unsigned int cols;  //storing columns of a matrix
    };

    //handlerror declaration : to display file and line numbers of erroneous lines
    static void HandleError( hipError_t err, const char *file, int line ) {
        if (err != hipSuccess) {
            cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
        }
    }

    //handle error alias name declaration
    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


    //global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
    //matrix multiplication is being done tile by tile
    __global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
    {   
        //shared memory takes one tile at a time
        __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];   //to store tiles for array 1
        __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];   //to store tiles for array 2

        //threads x and y index for the current block
        unsigned int tx=threadIdx.x;    
        unsigned int ty=threadIdx.y;

        unsigned int c=blockIdx.x*blockDim.x + threadIdx.x; //row value using x-index of current thread
        unsigned int r=blockIdx.y*blockDim.y + threadIdx.y; //column value using y-index of current thread

        unsigned int idx=c*rows1+r;             //column major index, using row and column value
        
        float val=0;        //register to store multiplication result initialized to zero

        for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)    //going over all tiles one by one, with each m
        {

            int var1=m*TILE_WIDTH+tx ;      //x thread value for current tile
            int var2=m*TILE_WIDTH+ty ;      //y thread value for current tile
            
            //copying a tile from array1
            if (r < rows1 && var1 < rows2)      //if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
                S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
            else
                    S1[ty][tx]=0;                   //storing zero, since there is no valid value
                __syncthreads();                        //syncing all threads once shared memory S1 is stored
            
            //copying a tile from array2
                if(c < cols2 && var2 < rows2)   //if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
                    S2[ty][tx]=array2[var2+rows2*c];    //storing the valid value
                else 
                    S2[ty][tx]=0;       //storing zero, since no valid value
            __syncthreads();        //synchronizing threads
            

            for(int i=0; i<TILE_WIDTH;i++)  //going over entire tile, ty row in S1 and tx column in S2
                val+=S1[ty][i]*S2[i][tx];   //and multiplying elements
            __syncthreads();        //synchronizing threads

        }
        
        if(r < rows1 && c< cols2)   //removing degenerate cases
            array3[idx]=val;    //saving multiplication result to global memory
            
    }

    float* gemm(float *array_A, float *array_B, int M_Arows, int M_Acols, int M_Brows, int M_Bcols) {

        float* array_C=(float*)malloc(M_Arows*M_Bcols*sizeof(float));//array to store gpu result in column major format
        
        float* array_D=(float*)malloc(M_Arows*M_Bcols*sizeof(float));//arary to store cublas result in column major format

        //GPU DEVICE PROPERTIES and selecting a GPU for calculation
        int nDevices;
        HANDLE_ERROR(hipGetDeviceCount(&nDevices));

        hipDeviceProp_t prop;
        HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));    //using GPU0

        //BLOCK AND GRID SIZE DECLARATION
        float thread_block=sqrt(prop.maxThreadsPerBlock);   //2D blocks used
        dim3 DimGrid(ceil(M_Bcols/thread_block),ceil(M_Arows/thread_block),1); //image saved as a 2D grid
        dim3 DimBlock(thread_block,thread_block,1);

        size_t Sbytes = 2* DimBlock.x * DimBlock.y ;    //2 arrays used in the calculation, hence 2 * DimBlock.x * DimBlock.y
        
        //Checking if sufficient shared memory available or not

        if(prop.sharedMemPerBlock < Sbytes){
            std::cout<<"ERROR: insufficient shared memory"<<std::endl;
            exit(1);
        }

        //GPU MEMORY ALLOCATION
        float *array_A_gpu, *array_B_gpu, *array_C_gpu, *array_D_gpu;   //gpu arrays declared
       
        HANDLE_ERROR(hipMalloc(&array_A_gpu,M_Arows*M_Acols*sizeof(float))); //allocate space to store arrayA

        HANDLE_ERROR(hipMalloc(&array_B_gpu,M_Brows*M_Bcols*sizeof(float))); //allocate space to store arrayB

        HANDLE_ERROR(hipMalloc(&array_C_gpu,M_Arows*M_Bcols*sizeof(float))); //allocate space to store gpu result

        HANDLE_ERROR(hipMalloc(&array_D_gpu,M_Arows*M_Bcols*sizeof(float))); //allocate space to store cublas result


        //COPY TO GPU MEMORY
        HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_Arows*M_Acols*sizeof(float), hipMemcpyHostToDevice));//copy arrayA to gpu

        HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_Brows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayB to gpu

        HANDLE_ERROR(hipMemcpy(array_C_gpu, array_C, M_Arows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayC to gpu

        HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_Arows*M_Bcols*sizeof(float), hipMemcpyHostToDevice));//copy arrayD to gpu


        //time measurement for matrix multiplication
        hipEvent_t start1, stop1;
        
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
        
        //MATRIX MULTIPLICATION USING KERNEL
        hipEventRecord(start1);
        matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_Arows,M_Acols,array_B_gpu,M_Brows,M_Bcols,array_C_gpu);//calling the kernel
        hipEventRecord(stop1);

        hipEventSynchronize(stop1);

        float milliseconds1 = 0, milliseconds2 = 0;//storing the execution time in milliseconds
        
        hipEventElapsedTime(&milliseconds1, start1, stop1);//get the time in milliseconds
        cout<<"Time taken by GPU GEMM = "<<milliseconds1<<" ms"<<endl;//printing time taken by GPU

        //copy to CPU MEMORY
        HANDLE_ERROR(hipMemcpy(array_C, array_C_gpu, M_Arows*M_Bcols*sizeof(float), hipMemcpyDeviceToHost));//copying result of multiplication from gpu to cpu

        //Creating handle for CUBLAS
        hipblasHandle_t handle;
        hipblasCreate(&handle);  

        //parameter declaration for cublas implementation
        float alpha = 1.0;
        float beta = 0.0;
        
        //cublas time measurement
        hipEvent_t start2, stop2;
        
        hipEventCreate(&start2);
        hipEventCreate(&stop2);

        //MATRIX MULTIPLICATION USING CUBLAS 
        hipEventRecord(start2);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_Arows, M_Bcols, M_Acols, &alpha, array_A_gpu, M_Arows, array_B_gpu, M_Brows, &beta, array_D_gpu, M_Arows);
        hipEventRecord(stop2);

        hipEventSynchronize(stop2);

        hipEventElapsedTime(&milliseconds2, start2, stop2);//get the time in milliseconds
        cout<<"Time taken by CUBLAS= "<<milliseconds2<<" ms"<<endl;//printing time taken by CUBLAS
        
        //copy to CPU MEMORY
        
            HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_Arows*M_Bcols*sizeof(float), hipMemcpyDeviceToHost)); 
//copy result of multiplication using CUBLAS from gpu to cpu

        //CALCULATING MEAN SQUARED ERROR IN BOTH METHODS OF MATRIX MULTIPLICATION
        float mse=0; //mean squared error;

     /*   for(int i=0; i<M_Arows*M_Bcols;i++) {
            mse=mse+(array_C[i]-array_D[i])*(array_C[i]-array_D[i]);//calculating element by element
           // printf("%.3f ", array_C[i]);
        }
        mse=mse/(M_Arows*M_Bcols);  //taking the mean of squared error
            
        cout<<endl<<"Mean square error = "<<mse<<endl;//printing out the mean squared error*/
	return array_C;

    }
float* matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2)
{
	float* C=(float*)malloc(rows1*cols2*sizeof(float));
	
	//initailize the array to zero
	for(int idx=0; idx<rows1*cols2;idx++)
	{
		C[idx]=0;
		int c=(int)(idx/rows1);
		int r=idx%rows1;

		for(int k=0;k<rows2;k++)
		{
			C[idx]+=array1[rows1*k+r]*array2[rows2*c+k];
		}

	}	
	
	return C;

}


float* flatten_kernel(float * weights, int k, int d, int c_rows){
    int c_cols = (k + (k-1)*(d-1))*(k + (k-1)*(d-1));
    float *canvas=(float*)calloc(C*c_rows*c_cols,sizeof(float));
    int itr = 0;
    int k_id = 0;
    for(int dilation = 1; dilation<=d; dilation++){
    int cur_kernel_size = k + (k-1)*(dilation-1);
    for(int kernel_id = 0; kernel_id < c_rows/4; kernel_id++){



	
    	itr =    k_id*c_cols*C + (d-dilation)* pow(c_cols,0.5) + (d - dilation);

	//cout<< itr <<"  itr after 1 filter!!!!!!!!!!"<<k<<" k "<<C<<" c \n";
    for(int weight_id = 0; weight_id < k*k*C; weight_id++){
    canvas[itr] = weights[k_id*k*k*C + weight_id];
   /* cout<< weights[k_id*k*k+weight_id]<<" WEIGHTS"<<endl;
    cout<<"  k_id  "<<k_id<<endl;
    cout<<k_id*k*k*C + weight_id<<"    index    "<<endl;
    cout<<canvas[itr]<<" CANVAS "<<itr <<endl;*/
    itr++;
//cout<<"weight id ************************ === "<<weight_id<< "        itr =  "<<itr<<endl;	
if((weight_id+1)%(k*k)==0){
                itr= kernel_id*c_cols*C  + (dilation-1)*(c_rows/4)*c_cols*C +  ((weight_id+1)/(k*k))*c_cols + (d-dilation)* pow(c_cols,0.5) + (d - dilation);
//if(dilation == 1 || dilation == 2)                
//cout<<" jump idx" <<kernel_id*c_cols*C  +  (dilation-1)*(c_rows/4)*c_cols*C +  ((weight_id+1)/(k*k))*c_cols<<"  +++++++++++##########weight id#######++++++++  "<<weight_id<< "    offset itr "<< itr <<endl;
continue; 
       }	
    if(((k_id*k*k*C + weight_id)+1)%(k)==0){
    	for(int last_col_pads = 0; last_col_pads<(dilation-1)*(pow(c_cols,0.5)) + (pow(c_cols,0.5)-(cur_kernel_size  ));last_col_pads++ ){
    		//canvas[itr] = 0;
    		//cout<<itr<< "TEST" <<endl;
    		itr++;
    	
    	}
    }
    else{
    	
    	for(int inner_cols = 0; inner_cols<(dilation-1);inner_cols++ ){
    		//canvas[itr] = 0;
    		itr++;
    	}
	

    }


    }
    k_id++;
    }
    }

/*	printf("\nROW MAJOR\n");
	for(int i=0; i < c_rows; i++){
		for(int j=0; j <C*c_cols; j++){
			int idx=(i*C*c_cols)+j;
			printf("%f ",canvas[idx]);
			if((idx+1)%(C*c_cols)==0){
				printf("\n\n");
			}
		}
	}
*/
    float *canvas_col=(float*)calloc(C*c_rows*c_cols,sizeof(float));
    itr=0;
    for(int i=0; i<C*c_cols;i++){
    for(int j=0; j < c_rows; j++) {
    canvas_col[itr]=canvas[(j*c_cols*C)+i];
    itr++;
    }
    }

return canvas_col;
	
}
void im2colOnHost(unsigned int n, float *matAc, float *matA, int H_, int W_, int L, int M, int K, int C)
{
   // Using grid-stride loop if too big problem size.
    // https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int idx = 0;
         idx < n;
         idx += 1)
    {
        int m = (idx/C ) / L;
        int l = (idx/C ) % L;
        int r = idx % C;

        // For each spatial position in output...
        if (m < M) {
            int w = m ;
            if (l < L) {
                int h = l;
                // For each kernel weight...
                for (int x = 0; x < K; x++) {

            h = l;
                    for (int y = 0; y < K; y++) {
                        if (r < C) {
                            matAc[((idx/C)*K*K*C)+(r*K*K) + (x*K + y)] = matA[(r*(H_*W_)) + w*W_ + h];
            h++;
}
                    }w++;
                }
            }
        }
    }
}
 
// DEVICE KERNEL
// Takes matrix A [float *matA] and transforms it
// into column representation [float *matAc] on GPU
__global__ 
void im2colOnDevice(unsigned int n, float *matAc, float *matA, int H_, int W_, int L, int M, int K, int C)
{
   // Using grid-stride loop if too big problem size.
    // https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    /*for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < n; 
         idx += blockDim.x * gridDim.x) 
    {*/
        int m = (idx/C ) / L;
        int l = (idx/C ) % L;
        int r = idx % C;
        
        // For each spatial position in output...
        if (m < M) {
            int w = m ;
            if (l < L) {
                int h = l;
                // For each kernel weight...
                for (int x = 0; x < K; x++) {
        
            h = l;
                    for (int y = 0; y < K; y++) {
                        if (r < C) {
                            matAc[((idx/C)*K*K*C)+(r*K*K) + (x*K + y)] = matA[(r*(H_*W_)) + w*W_ + h]; 
            h++;                        
}
                    }w++;
                }
            }
        }
    //}
}
 
float* padding(unsigned int maxDilation, float* matInput) {
    int newH=H+(2*maxDilation);
    int newW=W+(2*maxDilation);
    //const size_t size\ = newH*newW*sizeof(float);
    float *paddedInput=(float *)calloc(newH*newW, sizeof(float));
    for(int x=0; x<H;x++) {
        for(int y=0; y<W;y++) {
            paddedInput[((x+maxDilation)*newW)+y+maxDilation]=matInput[(x*W) + y];
        }

    }
    return paddedInput;

}

float* paddingFile(unsigned int maxDilation, char * fileName) {
    int newH=H+(2*maxDilation);
    int newW=W+(2*maxDilation);

    ifstream infile;
    infile.open(fileName);

    //const size_t size\ = newH*newW*sizeof(float);
    float *paddedInput=(float *)calloc(newH*newW*C, sizeof(float));
for(int c=0; c<C; c++) {
    for(int x=0; x<H;x++) {
        for(int y=0; y<W;y++) {
           infile >>  paddedInput[(c*newH*newW) + (((x+maxDilation)*newW)+y+maxDilation)];
        }

    }
}
    return paddedInput;

}



float round6(float var)
{
    // we use array of chars to store number
    // as a string.
    char str[40];
 
    // Print in string the value of var
    // with two decimal point
    sprintf(str, "%.6f", var);
 
    // scan string value in var
    sscanf(str, "%f", &var);
 
    return var;
}
 



void program(unsigned int blockSize, unsigned int gridSize = 0)
{
    // CONSTS AND VARIABLES
    // Input/kernel/output counts and sizes
    const unsigned int countA = H*W*C;
    const size_t sizeA = countA*sizeof(float);

    const unsigned int countF = K*K*C;
    const size_t sizeF = countF*sizeof(float);
    int paddedH=H+(2*maxDilation);
    int paddedW=W+(2*maxDilation);    
    const unsigned int L = H;
    const unsigned int M = W;
    const unsigned int KERNELS=L*M*C;	
    
    //dilated kernel size
    int K_= K + (K-1)*(maxDilation-1);
    const unsigned int countF_ = K_*K_*C;
    const unsigned int countLR = L * M;
    const unsigned int countAc = countF_ * countLR;
    const size_t sizeAc = countAc*sizeof(float);
    //LOG("[i] INPUT IN COL PARAMS: %u elems, %u bytes\n", countAc, sizeAc);
	
    const unsigned int countKc = K_*K_*C*C_out;
    
    // PREPARE DATA

    
    char * fileName = (char*) malloc(13 * sizeof(char));
    sprintf(fileName, "in_%d_%d_%d_%d", H, W, C, C_out);
    
    float *matInput=paddingFile(maxDilation, fileName);
    // Alloc memory and copy data to device

    ifstream infile;
    char * fileNameW = (char*) malloc(13 * sizeof(char));
    sprintf(fileNameW, "weights_%d_%d_%d_%d", H, W, C, C_out);
    infile.open(fileNameW);
    float *matFlatten = (float *)malloc(sizeF*C_out);
    printf("KERNEL SIZE %d\n", countF);
    for (int i = 0; i < countF*C_out; i++) {
            infile>>matFlatten[i];
        }

    struct timeval start, end;
    gettimeofday(&start, NULL);
    float *devA, *devAc, *retAc, *retCpu;
    const size_t sizeI = C*paddedW*paddedH*sizeof(float);
    hipMalloc((void**)&devA, sizeI); 
    hipMalloc((void**)&devAc, sizeAc); 
    retAc = (float *)malloc(sizeAc);
	
    hipMemcpy(devA, matInput, sizeI, hipMemcpyHostToDevice); 

    // Compute default grid size if it wasn't passed
    const unsigned int KERNELS_NUM = L * M * C;
    if (gridSize == 0)
        gridSize = (KERNELS_NUM + blockSize - 1) / blockSize;
     
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  
    float thread_block=sqrt(prop.maxThreadsPerBlock);
    unsigned int GRID_SIZE = (KERNELS + thread_block - 1) / thread_block;
        // Run im2col computation on device and copy results
        struct timeval t3, t4;
        gettimeofday(&t3, NULL);
        im2colOnDevice<<<GRID_SIZE, thread_block>>>(KERNELS, devAc, devA, paddedH, paddedW, L, M, K_ , C);
    	    gettimeofday(&t4, NULL);
        LOG("  [!] FINISHED CALCULATING im2col ON DEVICE %.16fms\n",(t4.tv_usec-t3.tv_usec)/1000.0+(t4.tv_sec-t3.tv_sec)*1000.0);    
        hipMemcpy(retAc, devAc, sizeAc, hipMemcpyDeviceToHost);


    //GEMM
    
    struct timeval flattens, flattene;
    gettimeofday(&flattens, NULL);
    float* kernelMatrix=flatten_kernel(matFlatten,K, maxDilation, C_out);
    gettimeofday(&flattene, NULL);
     LOG("  [!] FINISHED CALCULATING Flatten ON DEVICE %.16fms\n",(flattene.tv_usec-flattens.tv_usec)/1000.0+(flattene.tv_sec-flattens.tv_sec)*1000.0);
    float *res_gemm = gemm(kernelMatrix, retAc, C_out, countF_, K_ *K_ *C, countLR);
    gettimeofday(&t4, NULL);
    LOG("  [!] FINISHED CALCULATING CoConv ON DEVICE %.16fms\n",(t4.tv_usec-t3.tv_usec)/1000.0+(t4.tv_sec-t3.tv_sec)*1000.0);

    //CPU

    struct timeval cpu_start, cpu_end;
    gettimeofday(&cpu_start, NULL);
    retCpu = (float *)malloc(sizeAc);
    im2colOnHost(KERNELS, retCpu, matInput, paddedH, paddedW, L, M, K_ , C);
    float *res_gemm_cpu = matrix_mult(kernelMatrix, C_out, countF_, retAc, K_*K_*C, countLR);
    gettimeofday(&cpu_end, NULL);
    LOG("  [!] FINISHED CALCULATING CoConv ON CPU %.16fms\n",(cpu_end.tv_usec-cpu_start.tv_usec)/1000.0+(cpu_end.tv_sec-cpu_start.tv_sec)*1000.0);


    ifstream output;
    output.open("op");
    float mse=0.0;
    gettimeofday(&end, NULL);
     LOG("  [!] FINISHED CALCULATING ON DEVICE %.16fms\n",(end.tv_usec-start.tv_usec)/1000.0+(end.tv_sec-start.tv_sec)*1000.0);
    char * fileN = (char*) malloc(13 * sizeof(char));
        sprintf(fileN, "out.txt");
        FILE * fp;
        fp = fopen(fileN,"w");
    for(int c=0; c<C_out; c++){
    int spaces=0;
    int count=0;
    for (int i = 0; i < countLR; i++) {
    spaces++;
    int idx=i*C_out  + (c);

    float o =0.0;
            output >> o;
            mse += (round6(o)-res_gemm[idx])*(round6(o)-res_gemm[idx]);
    	fprintf(fp, "%f\n", res_gemm[idx]);
        }
    }
     fclose(fp);
    mse/=countLR*C_out;
    printf("\n MSE: %f", mse);
        // CLEAN UP
        hipFree(devA);
        hipFree(devAc);
        
        //free(matA);
        free(matInput);
        free(matFlatten);
        free(retAc);
        free(res_gemm);
    }

int main(int argc, char * argv[])
{
    // Enforce default block and grid sizes
    unsigned int blockSize = 256;
    unsigned int gridSize = 0;

    // Calculate max needed kernels/threads number
    const unsigned int L = H;
    const unsigned int M = W;
    const unsigned int KERNELS_NUM = L * M * C;

    // Prepare variables for time measurement
    struct timeval t1, t2;
    double elapsedTime, totalTime = 0;
    int totalRuns = 1;

    H = (unsigned int) atoi(argv[1]);
    W = (unsigned int) atoi(argv[2]);
    C = (unsigned int) atoi(argv[3]);
    C_out = (unsigned int) atoi(argv[4]);
    
    // First warm-up run
   // LOG("--------- WARM-UP ---------\n");
    //program(256);
    //LOG("--------- WARM-UP ---------\n\n");

#ifdef PERFTEST
    // Average over 10 runs
    totalRuns = 1;
    
    // Open file for perf logs
    std::fstream fperflog("perflog.csv", std::ios::out);
    if (fperflog.good())
    {
        // Measure effect of different block sizes
        const unsigned int MAX_BLOCK_SIZE = 2048;
        for (blockSize = 32; blockSize <= 34; blockSize *= 2) {
            const unsigned int MAX_GRID_SIZE = (KERNELS_NUM + blockSize - 1) / blockSize;
            LOG("  [!] For %d blocks, max grid size is %d\n", blockSize, MAX_GRID_SIZE);
            for (gridSize = 1; gridSize <=1; gridSize *= 2) {
                if (gridSize <= MAX_GRID_SIZE) {
                    totalTime = 0;
                    for (int i = 0; i < 1; i++)
#endif
                    {
                        // Start timer
                        gettimeofday(&t1, NULL);
                    
                       // WORK HARD!
                        program(blockSize, gridSize);
                    
                        // Stop timer
                        gettimeofday(&t2, NULL);
                    
                        // Compute the elapsed time in millisec
                        elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
                        elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
                    
                        totalTime += elapsedTime;
                    }
                    LOG("  [!] Whole program took %.3fms averaged over %d runs\n", totalTime / totalRuns, totalRuns);
#ifdef PERFTEST
                    fperflog << blockSize << "," << gridSize << "," << elapsedTime << std::endl;
                } else {
                    // Meaningless data, there is more grids ten data cat utilize 
                    fperflog << blockSize << "," << gridSize << "," << -1 << std::endl;
                }
            }
        }
        
        // Close file
        fperflog.close();
    }
#endif

    return EXIT_SUCCESS;
}




